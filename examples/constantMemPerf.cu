#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <sstream>
#include <vector>
#include <string>
#include <chrono>

#include "umpire/ResourceManager.hpp"
#include <hip/hip_runtime_api.h>

__global__ void add_constant_kernel(const double *a, double *d, double *c, int size)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < size)
  {
    c[i] = a[i] + d[i];
  }
}

__global__ void add_kernel(const double *a, const double *b, double *c, int size)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < size)
    c[i] = a[i] + b[i];
}

void check_error(void)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}

int main(int, char**)
{
  // auto& rm = umpire::ResourceManager::getInstance();

  const int size = 1024 * 8;
  const int run_times = 10000;
  double timing_device = 0;
  double timing_const = 0;

  auto& rm = umpire::ResourceManager::getInstance();

  std::cout << "Available allocators: ";
  for (auto s : rm.getAvailableAllocators()){
    std::cout << s << "  ";
  }
  std::cout << std::endl;

  // double *a = (double*)malloc(sizeof(double) * size);
  // double *b = (double*)malloc(sizeof(double) * size);
  // double *sum = (double*)malloc(sizeof(double) * size);

  auto host_alloc = rm.getAllocator("HOST");
  double *sum = static_cast<double*>(host_alloc.allocate(size*sizeof(double)));
  double *a = static_cast<double*>(host_alloc.allocate(size*sizeof(double)));
  double *b = static_cast<double*>(host_alloc.allocate(size*sizeof(double)));
  std::cout << "Host memory allocation finished\n";

  // double *d_a;
  // double *d_b;
  // double *d_sum;

  auto dev_alloc = rm.getAllocator("DEVICE");
  double *d_sum = static_cast<double*>(dev_alloc.allocate(size*sizeof(double)));
  double *d_a = static_cast<double*>(dev_alloc.allocate(size*sizeof(double)));
  double *d_b = static_cast<double*>(dev_alloc.allocate(size*sizeof(double)));
  std::cout << "Device memory allocation finished\n";


  auto dev_const_alloc = rm.getAllocator("DEVICE_CONST");
  double *d_d = static_cast<double*>(dev_const_alloc.allocate(size*sizeof(double)));
  std::cout << "Device constant memory allocation finished\n";



  for (int i=0; i<size; i++)
  {
    a[i] = static_cast<double>(i);
    b[i] = a[i];
    sum[i] = 1;
  }

  // hipMemcpy(d_a, a, size*sizeof(double), hipMemcpyHostToDevice);
  rm.copy(d_a, a, size*sizeof(double));
  check_error();

    // hipMemcpy(d_b, b, size*sizeof(double), hipMemcpyHostToDevice);
  rm.copy(d_b, b, size*sizeof(double));
  check_error();

  // hipMemcpy(d_d, b, size*sizeof(double), hipMemcpyHostToDevice);
  rm.copy(d_d, b, size*sizeof(double));
  check_error();

  std::cout << "Memory copy finished\n";

  std::chrono::high_resolution_clock::time_point t1, t2;

// ----------------add kernel-------------------------
  for(int i=0; i<run_times; i++)
  {
    t1 = std::chrono::high_resolution_clock::now();
    add_kernel<<<256, 1024>>>(d_a, d_b, d_sum, size);
    t2 = std::chrono::high_resolution_clock::now();
    timing_device += std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count();
    check_error();
  }
  

  // hipMemcpy(sum, d_sum, size*sizeof(double), hipMemcpyDeviceToHost);
  rm.copy(sum, d_sum, size*sizeof(double));
  check_error();
  
  int final_sum = 0;
  for (int i=0; i<size; i++)
  {
    final_sum += sum[i];   
  }
  std::cout << "device memory result: " << final_sum << std::endl;
  std::cout << "time for device" << timing_device << std::endl;

// ----------------add const kernel-------------------------
  for(int i=0; i<run_times; i++)
  {
    t1 = std::chrono::high_resolution_clock::now();
    add_constant_kernel<<<2, 1024>>>(d_a, d_d, d_sum, size);
    t2 = std::chrono::high_resolution_clock::now();
    timing_const += std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count();
    check_error();
  }
  // hipMemcpy(sum, d_sum, size*sizeof(double), hipMemcpyDeviceToHost);
  rm.copy(sum, d_sum, size*sizeof(double));
  check_error();

  int final_sum_const = 0;
  for (int i=0; i<size; i++)
  {
     final_sum_const += sum[i];
  }
  std::cout << "const memory result:  " << final_sum_const << std::endl;
  std::cout << "time for const" << timing_const << std::endl;

  if (final_sum_const - final_sum == 0)
    std::cout << "Same result, success" << std::endl;
  else
    std::cout << "Wrong result" << std::endl;

  // hipFree(d_a);
  // hipFree(d_b);
  // hipFree(d_sum);
  // free(a);
  // free(b);
  // free(sum);

  dev_alloc.deallocate(d_a);
  dev_alloc.deallocate(d_b);
  dev_alloc.deallocate(d_sum);
  host_alloc.deallocate(a);
  host_alloc.deallocate(b);
  host_alloc.deallocate(sum);
  dev_const_alloc.deallocate(d_d);

  return 0;
}