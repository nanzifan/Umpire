#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory
//
// Created by Zifan Nan, nan1@llnl.gov
// LLNL-CODE-747640
//
// All rights reserved.
//
// This file is part of Umpire.
//
// For details, see https://github.com/LLNL/Umpire
// Please also see the LICENSE file for MIT license.
//////////////////////////////////////////////////////////////////////////////
// #ifndef UMPIRE_DefaultMemoryResource_INL
// #define UMPIRE_DefaultMemoryResource_INL

#include "umpire/resource/ConstantMemoryResource.hpp"
#include "umpire/ResourceManager.hpp"
#include "umpire/util/Macros.hpp"

#include <memory>
#include <sstream>

namespace umpire {
namespace resource {

// template<typename _allocator>
ConstantMemoryResource::ConstantMemoryResource(Platform platform, const std::string& name, int id) :
  MemoryResource(name, id),
  // m_allocator(),
  m_current_size(0l),
  m_highwatermark(0l),
  m_platform(platform)
{
  offset = 0;
  std::cout << "new ConstantMemoryResource" << std::endl;
}

// template<typename _allocator>
void* ConstantMemoryResource::allocate(size_t bytes)
{
  // void* ptr = m_allocator.allocate(bytes);

  void* ptr = nullptr;
  hipError_t error = ::hipGetSymbolAddress((void**)&ptr, umpire_internal_device_constant_memory);

  char* new_ptr = (char*)ptr + offset;
  offset += bytes;

  if (offset > 1024 * 64)
  {
    UMPIRE_LOG(Debug, "ask bytes more than max constant memory size (64KB), current size is " << offset - bytes << "bytes");
  }

  ResourceManager::getInstance().registerAllocation((void*)new_ptr, new util::AllocationRecord{ptr, bytes, this->shared_from_this()});

  m_current_size += bytes;
  if (m_current_size > m_highwatermark)
    m_highwatermark = m_current_size;

  UMPIRE_LOG(Debug, "(bytes=" << bytes << ") returning " << ptr);

  return (void*)new_ptr;
}

// template<typename _allocator>
void ConstantMemoryResource::deallocate(void* ptr)
{
  UMPIRE_LOG(Debug, "(ptr=" << ptr << ")");

  // m_allocator.deallocate(ptr);
  util::AllocationRecord* record = ResourceManager::getInstance().deregisterAllocation(ptr);
  m_current_size -= record->m_size;
  delete record;
}

// template<typename _allocator>
long ConstantMemoryResource::getCurrentSize()
{
  UMPIRE_LOG(Debug, "() returning " << m_current_size);
  return m_current_size;
}

// template<typename _allocator>
long ConstantMemoryResource::getHighWatermark()
{
  UMPIRE_LOG(Debug, "() returning " << m_highwatermark);
  return m_highwatermark;
}

// template<typename _allocator>
Platform ConstantMemoryResource::getPlatform()
{
  return m_platform;
}

} // end of namespace resource
} // end of namespace umpire
// #endif // UMPIRE_DefaultMemoryResource_INL
